//
// Created by pejman on 08/11/22.
//

#include "error.cuh"

namespace gpu
{

    [[gnu::noinline]] void _abortError(const char *msg, const char *fname, int line)
    {
        hipError_t err = hipGetLastError();
        if (err == hipError_t::hipSuccess)
            return;
        spdlog::error("{} ({}, line: {})", msg, fname, line);
        spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
        std::exit(1);
    }

}
