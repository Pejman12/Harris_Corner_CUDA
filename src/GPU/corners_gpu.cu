#include "hip/hip_runtime.h"
#include "corners_gpu.cuh"
#include "separable_convolution.cuh"
#include "error.cuh"
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/remove.h>

namespace gpu
{

    __global__ void matmult(double *a, size_t aPitch, double *b, size_t bPitch, const int width,
                            const int height, double *res, size_t resPitch)
    {
        const auto x = blockDim.x * blockIdx.x + threadIdx.x;
        const auto y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x >= width || y >= height)
            return;

        const auto *a_line = (double *)((char *)a + y * aPitch);
        const auto *b_line = (double *)((char *)b + y * bPitch);
        auto *res_line = (double *)((char *)res + y * resPitch);
        res_line[x] = a_line[x] * b_line[x];
    }

    __global__ void compute_harris(double *Wx2, size_t Wx2Pitch, double *Wy2, size_t Wy2Pitch,
                                   double *Wxy, size_t WxyPitch, const int width, const int height,
                                   double *res, size_t resPitch)
    {
        const auto x = blockDim.x * blockIdx.x + threadIdx.x;
        const auto y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x >= width || y >= height)
            return;

        const auto *Wx2_line = (double *)((char *)Wx2 + y * Wx2Pitch);
        const auto *Wy2_line = (double *)((char *)Wy2 + y * Wy2Pitch);
        const auto *Wxy_line = (double *)((char *)Wxy + y * WxyPitch);
        auto *res_line = (double *)((char *)res + y * resPitch);
        double det = Wx2_line[x] * Wy2_line[x] - Wxy_line[x] * Wxy_line[x];
        double trace = Wx2_line[x] + Wy2_line[x];
        res_line[x] = det / (trace + EPS);
    }

    template <unsigned int radio>
    __global__ void DilationSharedStep2(double *src, size_t spitch, double *dst, size_t dpitch,
                                        int width, int height, int tile_w, int tile_h)
    {
        extern __shared__ double smem[];
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        int bx = blockIdx.x;
        int by = blockIdx.y;
        int x = bx * tile_w + tx;
        int y = by * tile_h + ty - radio;
        // auto smem_line = (double *)((char *)smem + ty * blockDim.x);
        smem[ty * blockDim.x + tx] = 0.f;
        __syncthreads();
        if (x >= width || y < 0 || y >= height)
        {
            return;
        }
        const auto src_line = (double *)((char *)src + y * spitch);
        smem[ty * blockDim.x + tx] = src_line[x];
        __syncthreads();
        if (y < (by * tile_h) || y >= ((by + 1) * tile_h))
        {
            return;
        }
        double *smem_thread = &smem[(ty - radio) * blockDim.x + tx];
        double val = smem_thread[0];
        for (int yy = 1; yy <= 2 * radio; yy++)
        {
            val = fmax(val, smem_thread[yy * blockDim.x]);
        }
        auto dst_line = (double *)((char *)dst + y * dpitch);
        dst_line[x] = val;
    }

    template <unsigned int radio>
    __global__ void DilationSharedStep1(double *src, size_t spitch, double *dst, size_t dpitch,
                                        int width, int height, int tile_w, int tile_h)
    {
        extern __shared__ double smem[];
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        int bx = blockIdx.x;
        int by = blockIdx.y;
        int x = bx * tile_w + tx - radio;
        int y = by * tile_h + ty;
        // auto smem_line = (double *)(smem + ty * blockDim.x);
        smem[ty * blockDim.x + tx] = 0.f;
        __syncthreads();
        if (x < 0 || x >= width || y >= height)
        {
            return;
        }
        const auto src_line = (double *)((char *)src + y * spitch);
        smem[ty * blockDim.x + tx] = src_line[x];
        __syncthreads();
        if (x < (bx * tile_w) || x >= ((bx + 1) * tile_w))
        {
            return;
        }
        double *smem_thread = &smem[ty * blockDim.x + tx - radio];
        double val = smem_thread[0];
        for (int xx = 1; xx <= 2 * radio; xx++)
        {
            val = fmax(val, smem_thread[xx]);
        }
        auto dst_line = (double *)((char *)dst + y * dpitch);
        dst_line[x] = val;
    }

    template <unsigned int radio>
    void DilationTwoStepsShared(double *src, size_t spitch, double *dst, size_t dpitch, int width,
                                int height)
    {
        double *temp;
        size_t tempPitch;
        if (hipMallocPitch(&temp, &tempPitch, width * sizeof(double), height) != hipSuccess)
            abortError("Fail buffer allocation");
        int tile_w = 640;
        int tile_h = 1;
        dim3 block2(tile_w + (2 * radio), tile_h);
        dim3 grid2(ceil((float)width / tile_w), ceil((float)height / tile_h));
        DilationSharedStep1<radio><<<grid2, block2, block2.y * block2.x * sizeof(double)>>>(
            src, spitch, temp, tempPitch, width, height, tile_w, tile_h);
        hipDeviceSynchronize();
        tile_w = 8;
        tile_h = 64;
        dim3 block3(tile_w, tile_h + (2 * radio));
        dim3 grid3(ceil((float)width / tile_w), ceil((float)height / tile_h));
        DilationSharedStep2<radio><<<grid3, block3, block3.y * block3.x * sizeof(double)>>>(
            temp, tempPitch, dst, dpitch, width, height, tile_w, tile_h);
        hipDeviceSynchronize();
        hipFree(temp);
    }

    __global__ void harris_mask(double *harris, size_t harrisPitch, double *dilate,
                                size_t dilatePitch, const int width, const int height, bool *mask,
                                size_t maskPitch, const double threshold)
    {
        const auto x = blockDim.x * blockIdx.x + threadIdx.x;
        const auto y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x >= width || y >= height)
            return;

        const auto *harris_line = (double *)((char *)harris + y * harrisPitch);
        const auto *dilate_line = (double *)((char *)dilate + y * dilatePitch);
        auto *mask_line = (bool *)((char *)mask + y * maskPitch);
        mask_line[x] = (harris_line[x] > threshold) & (dilate_line[x] == harris_line[x]);
    }

    __global__ void apply_mask_harris(double *harris, size_t harrisPitch, bool *mask, size_t maskPitch,
                                      const int width, const int height)
    {
        const auto x = blockDim.x * blockIdx.x + threadIdx.x;
        const auto y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x >= width || y >= height)
            return;

        auto *harris_line = (double *)((char *)harris + y * harrisPitch);
        const auto *mask_line = (bool *)((char *)mask + y * maskPitch);
        harris_line[x] = mask_line[x] ? harris_line[x] : 0;
    }

    __global__ void harris_to_kpts(double *harris, size_t harrisPitch, point *kpts, size_t kptsPitch,
                                   const int width, const int height)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x >= width || y >= height)
            return;

        const auto *harris_line = (double *)((char *)harris + y * harrisPitch);
        auto *kpts_line = (point *)((char *)kpts + y * kptsPitch);
        kpts_line[x].x = x;
        kpts_line[x].y = y;
        kpts_line[x].score = harris_line[x];
    }

    struct remove_null_kpts
    {
        __host__ __device__
            bool operator()(const point &kpt) const
        {
            return kpt.score == 0;
        }
    };

    struct sort_kpts
    {
        __host__ __device__
            bool operator()(const point &a, const point &b) const
        {
            return a.score < b.score;
        }
    };

    static double *to_matrix(unsigned char **buffer, const int width, const int height)
    {
        auto res = (double *)malloc(height * width * sizeof(double));

#pragma omp parallel for schedule(dynamic) shared(buffer, res, height, width) default(none)
        for (auto i = 0; i < height; ++i)
            for (auto j = 0; j < width; ++j)
                res[i * width + j] = buffer[i][j];

        return res;
    }

    point *find_corners_gpu(unsigned char **buffer_, const int width, const int height,
                            int *nb_kpts)
    {
        int bsize = 32;
        int w = std::ceil((float)width / bsize);
        int h = std::ceil((float)height / bsize);

        spdlog::debug("running kernel of size ({},{})", w, h);

        dim3 dimGrid(w, h);
        dim3 dimBlock(bsize, bsize);

        // Device image buffer
        double *devBuffer;
        size_t BufferPitch;
        if (hipMallocPitch(&devBuffer, &BufferPitch, width * sizeof(double), height)
            != hipSuccess)
            abortError("Fail buffer allocation");
        const auto buffer = to_matrix(buffer_, width, height);
        if (hipMemcpy2D(devBuffer, BufferPitch, buffer, width * sizeof(double),
                         width * sizeof(double), height, hipMemcpyHostToDevice)
            != hipSuccess)
            abortError("Fail buffer copy");
        free(buffer);

        setGaussianKernel();
        setDerivGaussianKernel();

        // Device Buffer
        double *devTempBuffer;
        size_t TempBufferPitch;
        if (hipMallocPitch(&devTempBuffer, &TempBufferPitch, width * sizeof(double), height)
            != hipSuccess)
            abortError("Fail buffer allocation");

        // Device Ix
        double *devIx;
        size_t IxPitch;
        if (hipMallocPitch(&devIx, &IxPitch, width * sizeof(double), height) != hipSuccess)
            abortError("Fail Ix allocation");

        convolutionRowsGPU(devTempBuffer, TempBufferPitch, devBuffer, BufferPitch, width, height, KernelType::GAUSSIAN);
        convolutionColumnsGPU(devIx, IxPitch, devTempBuffer, TempBufferPitch, width, height, KernelType::GAUSSIAN_DERIV);

        // Device Iy
        double *devIy;
        size_t IyPitch;
        if (hipMallocPitch(&devIy, &IyPitch, width * sizeof(double), height) != hipSuccess)
            abortError("Fail Iy allocation");
        convolutionRowsGPU(devTempBuffer, TempBufferPitch, devBuffer, BufferPitch, width, height, KernelType::GAUSSIAN_DERIV);
        convolutionColumnsGPU(devIy, IyPitch, devTempBuffer, TempBufferPitch, width, height, KernelType::GAUSSIAN);
        hipFree(devBuffer);

        // Device Ix2
        double *devIx2;
        size_t Ix2Pitch;
        if (hipMallocPitch(&devIx2, &Ix2Pitch, width * sizeof(double), height) != hipSuccess)
            abortError("Fail Ix2 allocation");
        matmult<<<dimGrid, dimBlock>>>(devIx, IxPitch, devIx, IxPitch, width, height, devIx2,
                                       Ix2Pitch);
        hipDeviceSynchronize();

        // Device Iy2
        double *devIy2;
        size_t Iy2Pitch;
        if (hipMallocPitch(&devIy2, &Iy2Pitch, width * sizeof(double), height) != hipSuccess)
            abortError("Fail Iy2 allocation");
        matmult<<<dimGrid, dimBlock>>>(devIy, IyPitch, devIy, IyPitch, width, height, devIy2,
                                       Iy2Pitch);
        hipDeviceSynchronize();

        // Device Ixy
        double *devIxy;
        size_t IxyPitch;
        if (hipMallocPitch(&devIxy, &IxyPitch, width * sizeof(double), height) != hipSuccess)
            abortError("Fail Ixy allocation");
        matmult<<<dimGrid, dimBlock>>>(devIx, IxPitch, devIy, IyPitch, width, height, devIxy,
                                       IxyPitch);
        hipDeviceSynchronize();
        hipFree(devIx);
        hipFree(devIy);

        // Device Wx2
        double *devWx2;
        size_t Wx2Pitch;
        if (hipMallocPitch(&devWx2, &Wx2Pitch, width * sizeof(double), height) != hipSuccess)
            abortError("Fail Wx2 allocation");
        convolutionRowsGPU(devTempBuffer, TempBufferPitch, devIx2, Ix2Pitch, width, height, KernelType::GAUSSIAN);
        convolutionColumnsGPU(devWx2, Wx2Pitch, devTempBuffer, TempBufferPitch, width, height, KernelType::GAUSSIAN);

        // Device Wy2
        double *devWy2;
        size_t Wy2Pitch;
        if (hipMallocPitch(&devWy2, &Wy2Pitch, width * sizeof(double), height) != hipSuccess)
            abortError("Fail Wy2 allocation");
        convolutionRowsGPU(devTempBuffer, TempBufferPitch, devIy2, Iy2Pitch, width, height, KernelType::GAUSSIAN);
        convolutionColumnsGPU(devWy2, Wy2Pitch, devTempBuffer, TempBufferPitch, width, height, KernelType::GAUSSIAN);

        // Device Wxy
        double *devWxy;
        size_t WxyPitch;
        if (hipMallocPitch(&devWxy, &WxyPitch, width * sizeof(double), height) != hipSuccess)
            abortError("Fail Wxy allocation");
        convolutionRowsGPU(devTempBuffer, TempBufferPitch, devIxy, IxyPitch, width, height, KernelType::GAUSSIAN);
        convolutionColumnsGPU(devWxy, WxyPitch, devTempBuffer, TempBufferPitch, width, height, KernelType::GAUSSIAN);
        hipFree(devIx2);
        hipFree(devIy2);
        hipFree(devIxy);
        hipFree(devTempBuffer);

        // Device Harris
        double *devHarris;
        size_t HarrisPitch;
        if (hipMallocPitch(&devHarris, &HarrisPitch, width * sizeof(double), height)
            != hipSuccess)
            abortError("Fail Harris allocation");
        compute_harris<<<dimGrid, dimBlock>>>(devWx2, Wx2Pitch, devWy2, Wy2Pitch, devWxy, WxyPitch,
                                              width, height, devHarris, HarrisPitch);
        hipDeviceSynchronize();
        hipFree(devWx2);
        hipFree(devWy2);
        hipFree(devWxy);

        thrust::device_vector<double> harris_vec(width * height);

        if (hipMemcpy2D(thrust::raw_pointer_cast(harris_vec.data()), width * sizeof(double),
                         devHarris, HarrisPitch, width * sizeof(double), height,
                         hipMemcpyDeviceToDevice)
            != hipSuccess)
            abortError("Fail FlattenHarris copy");

        const auto harris_minmax = thrust::minmax_element(harris_vec.begin(), harris_vec.end());
        auto harris_min = *harris_minmax.first;
        auto harris_max = *harris_minmax.second;
        const auto compute_threshold = harris_min + (harris_max - harris_min) * THRESHOLD;

        // Device HarrisDilate
        double *devHarrisDilate;
        size_t HarrisDilatePitch;
        if (hipMallocPitch(&devHarrisDilate, &HarrisDilatePitch, width * sizeof(double), height)
            != hipSuccess)
            abortError("Fail HarrisDilate allocation");
        DilationTwoStepsShared<DISTANCE>(devHarris, HarrisPitch, devHarrisDilate, HarrisDilatePitch,
                                         width, height);

        // Device HarrisMask
        bool *devHarrisMask;
        size_t HarrisMaskPitch;
        if (hipMallocPitch(&devHarrisMask, &HarrisMaskPitch, width * sizeof(bool), height)
            != hipSuccess)
            abortError("Fail HarrisMask allocation");
        harris_mask<<<dimGrid, dimBlock>>>(devHarris, HarrisPitch, devHarrisDilate,
                                           HarrisDilatePitch, width, height, devHarrisMask,
                                           HarrisMaskPitch, compute_threshold);
        hipDeviceSynchronize();
        hipFree(devHarrisDilate);

        apply_mask_harris<<<dimGrid, dimBlock>>>(devHarris, HarrisPitch, devHarrisMask, HarrisMaskPitch, width, height);
        hipDeviceSynchronize();
        hipFree(devHarrisMask);

        // Device Kpts
        point *devKpts;
        size_t KptsPitch;
        if (hipMallocPitch(&devKpts, &KptsPitch, width * sizeof(point), height) != hipSuccess)
            abortError("Fail Kpts allocation");
        harris_to_kpts<<<dimGrid, dimBlock>>>(devHarris, HarrisPitch, devKpts, KptsPitch, width, height);
        hipDeviceSynchronize();
        hipFree(devHarris);

        thrust::device_vector<point> kpts_vec(width * height);

        if (hipMemcpy2D(thrust::raw_pointer_cast(kpts_vec.data()), width * sizeof(point), devKpts, KptsPitch,
                         width * sizeof(point), height, hipMemcpyDeviceToDevice) != hipSuccess)
            abortError("Fail Kpts copy");
        hipFree(devKpts);

        remove_null_kpts remove_struct;
        sort_kpts sort_struct;

        const auto new_last = thrust::remove_if(kpts_vec.begin(), kpts_vec.end(), remove_struct);

        thrust::sort(kpts_vec.begin(), new_last, sort_struct);

        *nb_kpts = new_last - kpts_vec.begin();
        *nb_kpts = std::min(MAX_KPTS, *nb_kpts);
        auto *kpts = (point *)malloc(*nb_kpts * sizeof(point));
        if (hipMemcpy(kpts, thrust::raw_pointer_cast(kpts_vec.data()), *nb_kpts * sizeof(point), hipMemcpyDeviceToHost) != hipSuccess)
            abortError("Fail Kpts copy");

        return kpts;
    }
}